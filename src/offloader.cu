#include "offloader.h"
#include "host.h"
#include "timer.h"
#include "macro.h"
#include "bwa.h"
#include <locale.h>
#include "bwamem_GPU.cuh"
#include "batch_config.h"
#include "streams.cuh"
#include <future>
#include <iostream>
#include <iomanip>
#include <atomic>
#include <mutex>
#include <sys/time.h>
#include <stdlib.h>
#include <string.h>
#include <math.h>
#include <unistd.h>
#include <iostream>
#include <ostream>
#include <thread>

extern float tprof[MAX_NUM_GPUS][MAX_NUM_STEPS];

/**
 * @brief convert current host addresses on a minibatch's transfer_data to their (future) addresses on GPU
 * assuming name, seq, comment, qual pointers on trasnfer_data still points to host memory
 *
 * @param seqs
 * @param n_seqs
 * @param transfer_data transfer_data_t object where these reads reside
 */
void convert2DevAddr(transfer_data_t *transfer_data)
{
	auto reads = transfer_data->h_seqs;
	auto n_reads = transfer_data->n_seqs;
	auto first_read = reads[0];
	for (int i = 0; i < n_reads; i++)
	{
		reads[i].name = reads[i].name - first_read.name + transfer_data->d_seq_name_ptr;
		reads[i].seq = reads[i].seq - first_read.seq + transfer_data->d_seq_seq_ptr;
		reads[i].comment = reads[i].comment - first_read.comment + transfer_data->d_seq_comment_ptr;
		reads[i].qual = reads[i].qual - first_read.qual + transfer_data->d_seq_qual_ptr;
	}
}

/**
 * @brief copy a minibatch of n_reads from superbatch to transfer_data minibatch's pinned memory, starting from firstReadId. 
 * Read info are contiguous, but name, comment, seq, qual are not
 * 
 * @param superbatch_data
 * @param transfer_data 
 * @param firstReadId 
 * @param n_reads 
 */
void copyReads2PinnedMem(superbatch_data_t *superbatch_data, transfer_data_t *transfer_data, int firstReadId, int n_reads){
	int lastReadId = firstReadId + n_reads - 1; 
	// copy name, comment, seq, qual one by one
	for (int i = firstReadId; i <= lastReadId; i++){
		bseq1_t *read = &(superbatch_data->reads[i]);
		char *toAddr;

		toAddr = transfer_data->h_seq_seq_ptr + transfer_data->h_seq_seq_size;
		memcpy(toAddr, read->seq, read->l_seq + 1); // size + 1 for null-terminating char
		read->seq = toAddr;
		transfer_data->h_seq_seq_size += read->l_seq + 1;
	}
	// copy read info
	memcpy(transfer_data->h_seqs, &superbatch_data->reads[firstReadId], n_reads * sizeof(bseq1_t));
}

/**
 * @brief load a small batch from superbatch to transfer_data, up to MB_MAX_COUNT. 
 * Return number of reads loaded into transfer_data->n_seqs. return 0 if no read loaded
 * after loading, translate reads' addresses to GPU and transfer to GPU,
 * @param transfer_data
 * @param superbatch_data
 * @param num_loaded number of reads loaded from this superbatch before this minibatch
 * @return int number of reads loaded into transfer_data->n_seqs
 */
static void push(transfer_data_t *tran, superbatch_data_t *loaded_input,
        int *push_counter, std::mutex *push_m, g3_opt_t *g3_opt,
        int *actual_push_count,
        double *func_elapsed_ms)
{
    FUNC_TIMER_START;
    int push_count;
    int push_offset;

    push_m->lock();
    push_offset = *push_counter;

    if(push_offset >= loaded_input->n_reads){
        push_count = 0;
    } else{
        if(loaded_input->n_reads < g3_opt->batch_size){
            push_count = loaded_input->n_reads;
        } else{
            if(push_offset <= loaded_input->n_reads - g3_opt->batch_size){
                push_count = g3_opt->batch_size;
            } else if(push_offset < loaded_input->n_reads){
                push_count = loaded_input->n_reads - push_offset;
            } 
        }
    }

    *push_counter += push_count;
    push_m->unlock();

#ifdef VERBOSE
    if(push_count > 0){
        std::cerr << "Pushing input no. " << push_offset << " ~ no. "
            << push_offset + push_count - 1 << std::endl;
    }
#endif

    if(push_count == 0){
        tran->n_seqs = 0;
        *actual_push_count = 0;
        FUNC_TIMER_END;
        return;
    }

    // Push inputs to GPU device. ASSUMED MEMCPY DOES NOT FAIL
	resetTransfer(tran);
    tran->n_seqs = push_count;

	copyReads2PinnedMem(loaded_input, tran, push_offset, push_count);
	// at this point, all pointers on tran still point to name, seq, comment, qual addresses on loaded
	// translate reads' addresses to GPU addresses
	convert2DevAddr(tran);
	CUDATransferSeqsIn(tran);

    *actual_push_count = push_count;
    FUNC_TIMER_END;
    return;
}


/**
 * @brief output the previous batch of reads.
 * first transfer device's seqio to host's seq_io.
 * then write from host's seq_io to output
 * 
 * @param first_batch 
 * @param transfer_data 
 */

static void pull(transfer_data_t *tran, int *pull_counter, std::mutex *pull_m,
        int *actual_pull_count,
        double *func_elapsed_ms)
{
    FUNC_TIMER_START;
    int k = 0;
    int pull_offset;
    int pull_count = tran->n_seqs;

    if (pull_count==0){
        *actual_pull_count = 0;
        FUNC_TIMER_END;
        return;
    }

    pull_m->lock();
    pull_offset = *pull_counter;
    *pull_counter += pull_count;
    pull_m->unlock();

#ifdef VERBOSE
    std::cerr << "Pulling input no. " << pull_offset << " ~ no. "
        << pull_offset + pull_count - 1 << std::endl;
#endif

	//CUDATransferSamOut(tran); // FIXME

	// write from host's seq_io to output
#define BUFLEN 1024
	//bseq1_t *seqs = tran->h_seqs;
	for (int i = 0; i < pull_count; ++i){ // aggregate from memory then write to file
		//if (seqs[i].sam){
            //err_fputs(seqs[i].sam, stdout); // FIXME
            //pwrite(tran->fd_outfile, "Hello!", BUFLEN, pull_offset * BUFLEN);
            //write(tran->fd_outfile, "spider-man", 10);
        //} 
    }

    *actual_pull_count = pull_count;
    FUNC_TIMER_END;
    return;
}

static void deviceoffloader(int gpuid, superbatch_data_t *loadedinput,\
        process_data_t *batch_A, transfer_data_t *batch_B,\
        int *pull_counter, int *push_counter,\
        std::mutex *pull_m, std::mutex *push_m, g3_opt_t *g3_opt)
{
    int this_gpuid;
    if(hipSetDevice(gpuid) != hipSuccess){
       std::cerr << "Offloader for GPU no. " << gpuid 
           << " : hipSetDevice failed" << std::endl;
        return;
    }
    hipGetDevice(&this_gpuid);
    if(this_gpuid != gpuid){
       std::cerr << "Offloader for GPU no. " << gpuid 
           << " : hipSetDevice failed" << std::endl;
        return;
    } 

    double align_ms, pull_ms, push_ms;
    int pull_count, push_count, to_pull_count;
    while(true){
        std::thread t_align(bwa_align, gpuid, batch_A, g3_opt, &align_ms);
        to_pull_count = batch_B->n_seqs;
        pull(batch_B, pull_counter, pull_m, &pull_count, &pull_ms);
        push(batch_B, loadedinput, push_counter, push_m, g3_opt, 
                &push_count, &push_ms);
        t_align.join();

        std::cerr << "* GPU #" << gpuid << " | ";
        std::cerr << std::fixed << std::setprecision(2) << std::setw(8) 
            << align_ms;
        std::cerr << "ms | aligned " << batch_A->n_seqs << " reads" << std::endl;
        tprof[gpuid][ALIGN_TOTAL] += (float)align_ms;

        std::cerr << "* GPU #" << gpuid << " | ";
        std::cerr << std::fixed << std::setprecision(2) << std::setw(8) 
            << pull_ms;
        std::cerr << "ms | pulled " << pull_count << " / " << to_pull_count
            << " reads (prev. batch)" << std::endl;

        std::cerr << "* GPU #" << gpuid << " | ";
        std::cerr << std::fixed << std::setprecision(2) << std::setw(8) 
            << push_ms;
        std::cerr << "ms | pushed " << push_count << " reads (next batch)"
            << std::endl;

        swapData(batch_A, batch_B);
        if(batch_A->n_seqs == 0){
            to_pull_count = batch_B->n_seqs;
            pull(batch_B, pull_counter, pull_m, &pull_count, &pull_ms);

            std::cerr << "* GPU #" << gpuid << " | ";
            std::cerr << std::fixed << std::setprecision(2) << std::setw(8) 
                << pull_ms;
            std::cerr << "ms | pulled " << pull_count << " / " << to_pull_count
                << " reads (prev. FINAL batch)" << std::endl;
            break;
        }
    }

    resetTransfer(batch_B);
	return;
}

void offloader(superbatch_data_t *loadedinput, process_data_t *proc[MAX_NUM_GPUS],
        transfer_data_t *tran[MAX_NUM_GPUS], g3_opt_t *g3_opt,
        double *func_elapsed_ms)
{
    FUNC_TIMER_START;
    std::cerr << "* aligning " << loadedinput->n_reads << " reads with " 
        << g3_opt->num_use_gpus << " GPUs" << std::endl;
    if(loadedinput->n_reads == 0){
        FUNC_TIMER_END;
        return;
    }
    std::thread perGPU[MAX_NUM_GPUS];
    int pull_counter = 0;
    int push_counter = 0;
    std::mutex push_m, pull_m;

    int num_use_gpus = g3_opt->num_use_gpus;

    for(int j=0; j<num_use_gpus; j++){
        perGPU[j] = std::thread(deviceoffloader, j, loadedinput, proc[j], tran[j],\
                &pull_counter, &push_counter, &pull_m, &push_m, g3_opt);
    }
    for(int j=0; j<num_use_gpus; j++){
        perGPU[j].join();
    }

    FUNC_TIMER_END;
    return;
}
