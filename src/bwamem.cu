#include "hip/hip_runtime.h"
#include "streams.cuh"
#include "CUDAKernel_memmgnt.cuh"

#include "utils_CUDA.cuh"
#include "macro.h"
#include "timer.h"

#include "bwa.h"
#include "bwt_CUDA.cuh"
#include "bntseq.h"
#include "fastmap.h"

#include "hashKMerIndex.h"
#include "seed.cuh"
#include "preprocessing.cuh"
#include "aux.cuh"
#include "final_pack.h"

#include <string.h>
#include <fstream>
#include <iostream>

#include <hipcub/hipcub.hpp>
#include <thrust/device_vector.h>
#include <thrust/sort.h>


#define PRINT(LABEL) \
    g3_opt->print_mask & BIT(LABEL)

#define TIMER_INIT \
    hipEvent_t timer_event_start, timer_event_stop;\
    CUDA_CHECK(hipEventCreate(&timer_event_start));\
    CUDA_CHECK(hipEventCreate(&timer_event_stop));

#define TIMER_DESTROY \
    CUDA_CHECK(hipEventDestroy(timer_event_start));\
    CUDA_CHECK(hipEventDestroy(timer_event_stop));

#define TIMER_START(lap) \
    lap = 0;\
    CUDA_CHECK(hipEventRecord(timer_event_start, *(hipStream_t*)proc->CUDA_stream));\

#define TIMER_END(lap) \
    CUDA_CHECK(hipEventRecord(timer_event_stop, *(hipStream_t*)proc->CUDA_stream));\
    CUDA_CHECK(hipEventSynchronize(timer_event_stop));\
    CUDA_CHECK(hipEventElapsedTime(&lap, timer_event_start, timer_event_stop));\

#define LAUNCH_CHK(stream)\
{\
    hipStreamSynchronize(stream);\
    hipError_t err;\
    err = hipGetLastError();\
    if(err != hipSuccess)\
    {\
      fprintf(stderr,"GPU %d hipGetLastError(): %s %s %d\n", gpuid, hipGetErrorString(err), __FILE__, __LINE__);\
    }\
}


extern float tprof[MAX_NUM_GPUS][MAX_NUM_STEPS];

/*  main function for bwamem in GPU 
 *  return to seqs.sam
 *
 *
 * Stage    |   Substage        |   Step          |   note
 * ---------------------------------------------------------------------------
 * Seeding  |   SMEM seeding    |   seed          |   seeding core
 *          |   Reseeding       |   r2            |
 *          |                   |   r3            |
 * Chaining |   B-tree chaining |   sal           |
 *          |                   |   sort_seeds    |
 *          |                   |   chain         |   chaining core
 *          |                   |   sort_chains   |
 *          |                   |   filter        |
 * Extending|   Local extending |   pairgen       |
 *          |                   |   extend        |   extending core
 *          |                   |   filter_mark   |
 *          |                   |   sort_alns     |
 *          |   Traceback       |   pairgen       |
 *          |                   |   traceback     |   traceback core (w/ NM test)
 *          |                   |   finalize      |
 */
void bwa_align(int gpuid, process_data_t *proc, g3_opt_t *g3_opt,
        double *func_elapsed_ms)
{
    FUNC_TIMER_START;
    float step_lap;

    void *d_temp_storage;
    size_t temp_storage_size;
    int batch_size, num_seeds_to_extend;

    // Configure the GPU to use.
    int this_gpuid;
    CUDA_CHECK(hipSetDevice(gpuid));
    CUDA_CHECK(hipGetDevice(&this_gpuid));
    if(this_gpuid != gpuid){
        std::cerr << "hipSetDevice failed" << std::endl;
        exit(EXIT_FAILURE);
    } 

    // Initialize variables for aligning this batch of read sequences.
    if((batch_size = proc->batch_size) == 0){
        FUNC_TIMER_END;
        return;
    }
    CUDAResetBufferPool(proc->d_buffer_pools, g3_opt->batch_size);
    CUDA_CHECK(hipMemset(proc->d_Nseeds, 0, sizeof(int)));

    TIMER_INIT; // This should be placed after setting GPU.

    // (1/3) Seeding

    // SMEM seeding (Seeding 1/2)
    TIMER_START(step_lap);
    if(g3_opt->baseline){
        PREPROCESS_convert_bit_encoding_kernel 
            <<< batch_size, 32, 0, *(hipStream_t*)proc->CUDA_stream >>> (proc->d_seqs);
        LAUNCH_CHK(*(hipStream_t*)proc->CUDA_stream);

        MEMFINDING_collect_intv_kernel 
            <<< batch_size, 320, 512, *(hipStream_t*)proc->CUDA_stream >>> (
                    proc->d_opt, proc->d_bwt, proc->d_seqs,
                    proc->d_aux,	// output
                    proc->d_kmerHashTab,
                    proc->d_buffer_pools);
        LAUNCH_CHK(*(hipStream_t*)proc->CUDA_stream);

        filterSeeds <<< batch_size, WARPSIZE, 0, *(hipStream_t*)proc->CUDA_stream >>>(
                proc->d_opt, proc->d_aux);
        LAUNCH_CHK(*(hipStream_t*)proc->CUDA_stream);
    } else{
        preseedAndFilterV2 <<< batch_size, 320, 0, *(hipStream_t*)proc->CUDA_stream >>> (
                proc->d_fmIndex,
                proc->d_opt, proc->d_seqs, proc->d_aux, proc->d_kmerHashTab, proc->d_buffer_pools);
        LAUNCH_CHK(*(hipStream_t*)proc->CUDA_stream);
    }
    TIMER_END(step_lap);
    tprof[gpuid][S_SMEM] += step_lap;

    if(PRINT(_SMEM) || PRINT(_ALL_SEEDING)){
        for(int readID = 0; readID < batch_size; readID++)
        {
            printIntv<<<1, WARPSIZE, 0, *(hipStream_t*)proc->CUDA_stream>>>(proc->d_aux, readID, _SMEM);
            LAUNCH_CHK(*(hipStream_t*)proc->CUDA_stream);
        }
    }

    // Reseeding (Seeding 2/2)
    TIMER_START(step_lap);
    reseedV2 <<< ceil((float)batch_size / WARPSIZE) == 0 ? 1 
        : ceil((float)batch_size / WARPSIZE), WARPSIZE, 0, *(hipStream_t*)proc->CUDA_stream >>>(
                proc->d_fmIndex, proc->d_opt, proc->d_seqs, proc->d_aux, proc->d_kmerHashTab, 
                proc->d_buffer_pools, batch_size);
    LAUNCH_CHK(*(hipStream_t*)proc->CUDA_stream);

    TIMER_END(step_lap);
    tprof[gpuid][S_R2] += step_lap;

    TIMER_START(step_lap);
    reseedLastRound <<< ceil((float)batch_size / WARPSIZE) == 0 ? 1 
        : ceil((float)batch_size / WARPSIZE), WARPSIZE, 0, *(hipStream_t*)proc->CUDA_stream >>>(
                proc->d_fmIndex, proc->d_opt, proc->d_seqs, proc->d_aux, proc->d_kmerHashTab, batch_size);
    LAUNCH_CHK(*(hipStream_t*)proc->CUDA_stream);

    TIMER_END(step_lap);
    tprof[gpuid][S_R3] += step_lap;


    if(PRINT(_INTV) || PRINT(_ALL_SEEDING)){
        for(int readID = 0; readID < batch_size; readID++)
        {
            printIntv<<<1, WARPSIZE, 0, *(hipStream_t*)proc->CUDA_stream>>>(proc->d_aux, readID, _INTV);
            LAUNCH_CHK(*(hipStream_t*)proc->CUDA_stream);
        }
    }

#if 0
    // (2/3) Chaining

    // B-tree chaining (Chaining 1/1)
    TIMER_START(step_lap);
    saLookup <<< batch_size, 128, 0, *(hipStream_t*)proc->CUDA_stream >>> (
            proc->d_opt, proc->d_bwt, proc->d_bns, proc->d_seqs, proc->d_aux,
            proc->d_seq_seeds,
            proc->d_buffer_pools);
    LAUNCH_CHK(*(hipStream_t*)proc->CUDA_stream);

    TIMER_END(step_lap);
    tprof[gpuid][C_SAL] += step_lap;

    if(PRINT(_SEED) || PRINT(_ALL_CHAINING)){
        for(int readID = 0; readID < batch_size; readID++)
        {
            printSeed<<<1, WARPSIZE, 0, *(hipStream_t*)proc->CUDA_stream>>>(proc->d_seq_seeds, readID, _SEED);
            LAUNCH_CHK(*(hipStream_t*)proc->CUDA_stream);
        }
    }


    TIMER_START(step_lap);
    sortSeedsLowDim 
        <<< batch_size, SORTSEEDSLOW_BLOCKDIMX, 0, *(hipStream_t*)proc->CUDA_stream >>> (
                proc->d_seq_seeds,
                proc->d_buffer_pools);
    LAUNCH_CHK(*(hipStream_t*)proc->CUDA_stream);

    sortSeedsHighDim 
        <<< batch_size, SORTSEEDSHIGH_BLOCKDIMX, 0, *(hipStream_t*)proc->CUDA_stream >>> (
                proc->d_seq_seeds,
                proc->d_buffer_pools);
    LAUNCH_CHK(*(hipStream_t*)proc->CUDA_stream);

    TIMER_END(step_lap);
    tprof[gpuid][C_SORT_SEEDS] += step_lap;

    if(PRINT(_STSEED) || PRINT(_ALL_CHAINING)){
        for(int readID = 0; readID < batch_size; readID++)
        {
            printSeed<<<1, WARPSIZE, 0, *(hipStream_t*)proc->CUDA_stream>>>(proc->d_seq_seeds, readID, _STSEED);
            LAUNCH_CHK(*(hipStream_t*)proc->CUDA_stream);
        }
    }

    TIMER_START(step_lap);
    if(g3_opt->baseline){
#if 0
#define SEEDCHAINING_CHAIN_BLOCKDIMX 256
        SEEDCHAINING_chain_kernel <<< batch_size, SEEDCHAINING_CHAIN_BLOCKDIMX, 0, *(hipStream_t*)proc->CUDA_stream >>> (
                proc->d_opt, proc->d_bns, proc->d_seqs, proc->d_seq_seeds,
                proc->d_chains,	// output
                proc->d_buffer_pools);
#else
        BTreeChaining 
            <<< ceil((float)batch_size / WARPSIZE) == 0 ? 1 : ceil((float)batch_size / WARPSIZE), WARPSIZE, 0, *(hipStream_t*)proc->CUDA_stream >>>(
                    proc->d_opt, proc->d_bns, proc->d_seqs, proc->d_seq_seeds,
                    proc->d_chains,	// output
                    proc->d_buffer_pools);
#endif
    } else{
        BTreeChaining 
            <<< ceil((float)batch_size / WARPSIZE) == 0 ? 1 : ceil((float)batch_size / WARPSIZE), WARPSIZE, 0, *(hipStream_t*)proc->CUDA_stream >>>(

                    proc->d_opt, proc->d_bns, proc->d_seqs, proc->d_seq_seeds,
                    proc->d_chains,	// output
                    proc->d_buffer_pools);
    }
    LAUNCH_CHK(*(hipStream_t*)proc->CUDA_stream);

    TIMER_END(step_lap);
    tprof[gpuid][C_CHAIN] += step_lap;

    if(PRINT(_CHAIN) || PRINT(_ALL_CHAINING)){
        for(int readID = 0; readID < batch_size; readID++){
            printChain<<<1, WARPSIZE, 0, *(hipStream_t*)proc->CUDA_stream>>>(proc->d_chains, readID, _CHAIN);
            LAUNCH_CHK(*(hipStream_t*)proc->CUDA_stream);
        }
    }

    TIMER_START(step_lap);
    sortChainsDecreasingWeight 
        <<< batch_size, SORTCHAIN_BLOCKDIMX, 
        MAX_N_CHAIN*2*sizeof(uint16_t)+sizeof(mem_chain_t**), *(hipStream_t*)proc->CUDA_stream>>> 
            (proc->d_chains, proc->d_buffer_pools);
    LAUNCH_CHK(*(hipStream_t*)proc->CUDA_stream);

    TIMER_END(step_lap);
    tprof[gpuid][C_SORT_CHAINS] += step_lap;

    if(PRINT(_STCHAIN) || PRINT(_ALL_CHAINING)){
        for(int readID = 0; readID < batch_size; readID++)
        {
            printChain<<<1, WARPSIZE, 0, *(hipStream_t*)proc->CUDA_stream>>>(proc->d_chains, readID, _STCHAIN);
            LAUNCH_CHK(*(hipStream_t*)proc->CUDA_stream);
        }
    }

    TIMER_START(step_lap);
    CHAINFILTERING_filter_kernel <<< batch_size, CHAIN_FLT_BLOCKSIZE, MAX_N_CHAIN*(3*sizeof(uint16_t)+sizeof(uint8_t)), *(hipStream_t*)proc->CUDA_stream >>> (
            proc->d_opt, 
            proc->d_chains, 	// input and output
            proc->d_buffer_pools);
    LAUNCH_CHK(*(hipStream_t*)proc->CUDA_stream);

    TIMER_END(step_lap);
    tprof[gpuid][C_FILTER] += step_lap;

#if 0 // this code is anyway dead for short reads of our concern (<700bp).
    TIMER_START(lap);
    CHAINFILTERING_flt_chained_seeds_kernel <<< ceil((float)batch_size / WARPSIZE) == 0 ? 1 : ceil((float)batch_size / WARPSIZE), WARPSIZE, 0, *(hipStream_t*)proc->CUDA_stream >>>(
            proc->d_opt, proc->d_bns, proc->d_pac,
            proc->d_seqs, proc->d_chains, 	// input and output
            batch_size, proc->d_buffer_pools);
    TIMER_END(lap);
    step_lap += lap;
#endif

    if(PRINT(_FTCHAIN) || PRINT(_ALL_CHAINING)){
        for(int readID = 0; readID < batch_size; readID++)
        {
            printChain<<<1, WARPSIZE, 0, *(hipStream_t*)proc->CUDA_stream>>>(proc->d_chains, readID, _FTCHAIN);
            LAUNCH_CHK(*(hipStream_t*)proc->CUDA_stream);
        }
    }


    // (3/3) Extending

    // Extending -> Local extending (1/2)

    TIMER_START(step_lap);
    SWSeed <<< ceil((float)batch_size / WARPSIZE) == 0 ? 1 : ceil((float)batch_size / WARPSIZE), WARPSIZE, 0, *(hipStream_t*)proc->CUDA_stream >>>(
            proc->d_chains, proc->d_regs, proc->d_seed_records, proc->d_Nseeds, batch_size,
            proc->d_buffer_pools
           );
    LAUNCH_CHK(*(hipStream_t*)proc->CUDA_stream);

    CUDA_CHECK(hipMemcpy(&num_seeds_to_extend, proc->d_Nseeds, sizeof(int), hipMemcpyDeviceToHost));

    if(num_seeds_to_extend==0){
        proc->n_processed += proc->batch_size;
        //CUDA_CHECK(hipStreamSynchronize(*(hipStream_t*)proc->CUDA_stream));
        return;
    }

    ExtendingPairGenerate <<< ceil((float)num_seeds_to_extend/32.0), 32, 0, *(hipStream_t*)proc->CUDA_stream >>> (
            proc->d_opt, proc->d_bns, proc->d_pac, proc->d_seqs,
            proc->d_chains, proc->d_regs, proc->d_seed_records, proc->d_Nseeds,
            batch_size, proc->d_buffer_pools
           );
    LAUNCH_CHK(*(hipStream_t*)proc->CUDA_stream);

    TIMER_END(step_lap);
    tprof[gpuid][E_PAIRGEN] += step_lap;

    if(PRINT(_DETAIL)){
        std::cerr << "GPU " << gpuid << "  " << "# local extending pairs: " << num_seeds_to_extend << std::endl;
    }

    if(PRINT(_EXPAIR) || PRINT(_ALL_EXTENDING)){
        printPair<<<1, WARPSIZE, 0, *(hipStream_t*)proc->CUDA_stream>>>(proc->d_seed_records, num_seeds_to_extend, _EXPAIR);
        LAUNCH_CHK(*(hipStream_t*)proc->CUDA_stream);
    }


    TIMER_START(step_lap);
    if(g3_opt->baseline){
        localExtending_baseline <<< num_seeds_to_extend, WARPSIZE, 0, *(hipStream_t*)proc->CUDA_stream >>> (
                proc->d_opt,
                proc->d_chains, 		// input chains
                proc->d_seed_records,
                proc->d_regs,		// output array
                proc->d_Nseeds);
    } else{
        localExtending <<< num_seeds_to_extend, WARPSIZE, 0, *(hipStream_t*)proc->CUDA_stream >>> (
                proc->d_opt,
                proc->d_chains, 		// input chains
                proc->d_seed_records,
                proc->d_regs,		// output array
                proc->d_Nseeds);
        // TODO adopt agatha
    }
    LAUNCH_CHK(*(hipStream_t*)proc->CUDA_stream);

    TIMER_END(step_lap);
    tprof[gpuid][E_EXTEND] += step_lap;

    if(PRINT(_REGION) || PRINT(_ALL_EXTENDING)){
        for(int readID = 0; readID < batch_size; readID++)
        {
            printReg<<<1, WARPSIZE, 0, *(hipStream_t*)proc->CUDA_stream>>>(proc->d_regs, readID, _REGION);
            LAUNCH_CHK(*(hipStream_t*)proc->CUDA_stream);
        }
    }

    // remove duplicates
    TIMER_START(step_lap);
    filterRegions <<< batch_size, 320, 0, *(hipStream_t*)proc->CUDA_stream >>> (
            proc->d_opt, proc->d_bns,
            proc->d_chains, proc->d_regs, proc->d_buffer_pools
           );
    LAUNCH_CHK(*(hipStream_t*)proc->CUDA_stream);

    TIMER_END(step_lap);
    tprof[gpuid][E_FILTER_MARK] += step_lap;

    if(PRINT(_FTREGION) || PRINT(_ALL_EXTENDING)){
        for(int readID = 0; readID < batch_size; readID++)
        {
            printReg<<<1, WARPSIZE, 0, *(hipStream_t*)proc->CUDA_stream>>>(proc->d_regs, readID, _FTREGION);
            LAUNCH_CHK(*(hipStream_t*)proc->CUDA_stream);
        }
    }


    TIMER_START(step_lap);
    sortRegions <<< ceil((float)batch_size / WARPSIZE) == 0 ? 1 : ceil((float)batch_size / WARPSIZE), WARPSIZE, 0, *(hipStream_t*)proc->CUDA_stream >>>(
            proc->d_regs, batch_size, proc->d_buffer_pools);
    LAUNCH_CHK(*(hipStream_t*)proc->CUDA_stream);


    TIMER_END(step_lap);
    tprof[gpuid][E_SORT_ALNS] += step_lap;

    if(PRINT(_STREGION) || PRINT(_ALL_EXTENDING)){
        for(int readID = 0; readID < batch_size; readID++)
        {
            printReg<<<1, WARPSIZE, 0, *(hipStream_t*)proc->CUDA_stream>>>(proc->d_regs, readID, _STREGION);
            LAUNCH_CHK(*(hipStream_t*)proc->CUDA_stream);
        }
    }


    // Extending -> Traceback (2/2)

    TIMER_START(step_lap);
    CUDA_CHECK(hipMemset(proc->d_Nseeds, 0, sizeof(int)));

    FINALIZEALN_preprocessing1_kernel <<< ceil((float)batch_size / WARPSIZE) == 0 ? 1 : ceil((float)batch_size / WARPSIZE), WARPSIZE, 0, *(hipStream_t*)proc->CUDA_stream >>>(
            proc->d_regs, proc->d_alns, proc->d_seed_records, proc->d_Nseeds,
            proc->d_buffer_pools);
    LAUNCH_CHK(*(hipStream_t*)proc->CUDA_stream);

    int batch_num_alns;

    CUDA_CHECK(hipMemcpy(&batch_num_alns, proc->d_Nseeds, sizeof(int), hipMemcpyDeviceToHost));

    if(batch_num_alns==0){
        proc->n_processed += proc->batch_size;
        //CUDA_CHECK(hipStreamSynchronize(*(hipStream_t*)proc->CUDA_stream));
        FUNC_TIMER_END;
        return;
    }

    FINALIZEALN_preprocessing2_kernel <<< ceil((float)batch_num_alns / WARPSIZE) == 0 ? 1 : ceil((float)batch_num_alns / WARPSIZE), WARPSIZE, 0, *(hipStream_t*)proc->CUDA_stream >>>(
            proc->d_opt, proc->d_seqs,
            proc->d_pac, proc->d_bns,
            proc->d_regs, proc->d_alns, proc->d_seed_records, batch_num_alns,
            proc->d_sortkeys_in,	// sortkeys_in = bandwidth * rlen
            proc->d_seqIDs_in,
            proc->d_buffer_pools);
    LAUNCH_CHK(*(hipStream_t*)proc->CUDA_stream);


    // reverse query and target if aln position is on reverse strand
    FINALIZEALN_reverseSeq_kernel <<< batch_num_alns, WARPSIZE, 0, *(hipStream_t*)proc->CUDA_stream >>> (proc->d_seed_records, proc->d_alns, proc->d_buffer_pools);
    LAUNCH_CHK(*(hipStream_t*)proc->CUDA_stream);



    d_temp_storage = NULL;
    temp_storage_size = 0;
    // determine temporary storage requirement
    CUDA_CHECK(hipcub::DeviceRadixSort::SortPairs(d_temp_storage, temp_storage_size, proc->d_sortkeys_in, proc->d_sortkeys_out, proc->d_seqIDs_in, proc->d_seqIDs_out, batch_num_alns, 0, 8*sizeof(int), *(hipStream_t*)proc->CUDA_stream));
    // Allocate temporary storage
    CUDA_CHECK(hipMalloc(&d_temp_storage, temp_storage_size));
    // perform radix sort
    CUDA_CHECK(hipcub::DeviceRadixSort::SortPairsDescending(d_temp_storage, temp_storage_size, proc->d_sortkeys_in, proc->d_sortkeys_out, proc->d_seqIDs_in, proc->d_seqIDs_out, batch_num_alns, 0, 8*sizeof(int), *(hipStream_t*)proc->CUDA_stream));
    hipFree(d_temp_storage);

    TIMER_END(step_lap);
    tprof[gpuid][E_T_PAIRGEN] += step_lap;


    if(PRINT(_TBPAIR) || PRINT(_ALL_EXTENDING)){
        printPair<<<1, WARPSIZE, 0, *(hipStream_t*)proc->CUDA_stream>>>(proc->d_seed_records, batch_num_alns, _TBPAIR);
        LAUNCH_CHK(*(hipStream_t*)proc->CUDA_stream);
    }

    if(PRINT(_DETAIL)){
        std::cerr << "GPU " << gpuid << "  " << "# traceback pairs: " << batch_num_alns << std::endl;
    }


    TIMER_START(step_lap);
    if(g3_opt->baseline){
        traceback_baseline <<< ceil((float)batch_num_alns / WARPSIZE) == 0 ? 1 : ceil((float)batch_num_alns / WARPSIZE), WARPSIZE, 0, *(hipStream_t*)proc->CUDA_stream >>>(
                proc->d_opt,
                proc->d_seed_records, batch_num_alns, proc->d_alns, proc->d_seqIDs_out,
                proc->d_buffer_pools
               );
    } else{
        traceback<<< ceil((float)batch_num_alns / WARPSIZE) == 0 ? 1 : ceil((float)batch_num_alns / WARPSIZE), WARPSIZE, 0, *(hipStream_t*)proc->CUDA_stream >>>(
                proc->d_opt,
                proc->d_seed_records, batch_num_alns, proc->d_alns, proc->d_seqIDs_out,
                proc->d_buffer_pools
               );
    }
    LAUNCH_CHK(*(hipStream_t*)proc->CUDA_stream);

    TIMER_END(step_lap);
    tprof[gpuid][E_TRACEBACK] += step_lap;

    if(PRINT(_ALIGNMENT) || PRINT(_ALL_EXTENDING)){
        for(int readID = 0; readID < batch_size; readID++)
        {
            printAln<<<1, WARPSIZE, 0, *(hipStream_t*)proc->CUDA_stream>>>(proc->d_bns, proc->d_alns, readID, _ALIGNMENT);
            LAUNCH_CHK(*(hipStream_t*)proc->CUDA_stream);
        }
    }

    TIMER_START(step_lap);
    finalize<<< ceil((float)batch_num_alns / WARPSIZE) == 0 ? 1 : ceil((float)batch_num_alns / WARPSIZE), WARPSIZE, 0, *(hipStream_t*)proc->CUDA_stream >>>(
            proc->d_opt, proc->d_bns, proc->d_seqs,
            proc->d_regs, proc->d_alns, proc->d_seed_records, batch_num_alns,
            proc->d_buffer_pools
           );
    LAUNCH_CHK(*(hipStream_t*)proc->CUDA_stream);

    TIMER_END(step_lap);
    tprof[gpuid][E_FINALIZE] += step_lap;

    if(PRINT(_RESULT) || PRINT(_ALL_EXTENDING)){
        for(int readID = 0; readID < batch_size; readID++)
        {
            printAln<<<1, WARPSIZE, 0, *(hipStream_t*)proc->CUDA_stream>>>(proc->d_bns, proc->d_alns, readID, _RESULT);
            LAUNCH_CHK(*(hipStream_t*)proc->CUDA_stream);
        }
    }

    /*

    int *d_output_aln_offsets;   // indexed with readID
    int *d_output_cigar_offsets;    // index with alnID

    int *d_output_rids;    // indexed with alnID
    uint64_t *d_output_pos;    // indexed with alnID

    uint32_t *d_output_cigars;  // indexed with cigar offsets
        
    CUDA_CHECK(hipMalloc(&d_output_aln_offsets, sizeof(int) * batch_size));
    CUDA_CHECK(hipMalloc(&d_output_cigar_offsets, sizeof(int) * batch_num_alns));
    int batch_cigar_len;

    final_pack_compute_offsets(batch_size, proc->d_alns, batch_num_alns,
            d_output_aln_offsets, d_output_cigar_offsets, &batch_cigar_len);

    CUDA_CHECK(hipMalloc(&d_output_rids, sizeof(int) * batch_num_alns));
    CUDA_CHECK(hipMalloc(&d_output_pos, sizeof(uint64_t) * batch_num_alns));
    CUDA_CHECK(hipMalloc(&d_output_cigars, sizeof(uint32_t) * batch_cigar_len));

    final_pack_compact<<<NUM_BLOCKS, BLOCKDIM>>>(batch_size, proc->d_alns,
            d_output_aln_offsets, d_output_cigar_offsets,
            d_output_rids, d_output_pos, d_output_cigars);

    int rids[8];
    uint64_t pos[8];
    CUDA_CHECK(hipMemcpy(rids, d_output_rids, sizeof(int) * batch_num_alns,
                hipMemcpyDeviceToHost));
    CUDA_CHECK(hipMemcpy(pos, d_output_pos, sizeof(uint64_t) * batch_num_alns,
                hipMemcpyDeviceToHost));

    std::cerr << "rids: ";
    for(int k = 0; k < batch_num_alns; k++){
        std::cerr << rids[k] << (k == (batch_num_alns - 1) ? "\n" : ", ");
    }
    std::cerr << "pos: ";
    for(int k = 0; k < batch_num_alns; k++){
        std::cerr << pos[k] << (k == (batch_num_alns - 1) ? "\n" : ", ");
    }
    */

#endif

    if(PRINT(_BUFFER_USAGE)){
        printBufferInfoHost(proc->d_buffer_pools);
    }

    TIMER_DESTROY;
    FUNC_TIMER_END;
    return;
};
