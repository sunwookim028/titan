#include "hip/hip_runtime.h"
#include "host.h"
#include "timer.h"
#include "batch_config.h"
#include "offloader.h"
#include "streams.cuh"
#include <future>
#include <sys/time.h>
#include <stdlib.h>
#include <string.h>
#include <fstream>
#include <iostream>
#include <iomanip>
#include <climits>
#include <cfloat>
#include <thread>
using namespace std;

float tprof[MAX_NUM_GPUS][MAX_NUM_STEPS];

/**
 * @brief initiate memory for a super batch
 * @return superbatch_data_t*
 */
static superbatch_data_t *newSuperBatchData()
{
    superbatch_data_t *batch = (superbatch_data_t *)malloc(sizeof(superbatch_data_t));
    batch->n_reads = 0;
    // init memory for reads in the batch
    batch->reads = (bseq1_t *)malloc(SB_MAX_COUNT * sizeof(bseq1_t));
    batch->name = (char *)malloc(SB_NAME_LIMIT);
    batch->seqs = (char *)malloc(SB_SEQ_LIMIT);
    batch->comment = (char *)malloc(SB_COMMENT_LIMIT);
    batch->qual = (char *)malloc(SB_QUAL_LIMIT);
    if (batch->reads == nullptr || batch->name == nullptr || batch->seqs == nullptr || batch->comment == nullptr || batch->qual == nullptr)
    {
        fprintf(stderr, "[M::%-25s] can't malloc superbatch\n", __func__);
        exit(1);
    }

    batch->name_size = 0;
    batch->comment_size = 0;
    batch->seqs_size = 0;
    batch->qual_size = 0;

    if (bwa_verbose >= 3)
    {
        double nGB_allocated = (double)(SB_MAX_COUNT * sizeof(bseq1_t) + SB_NAME_LIMIT + SB_SEQ_LIMIT + SB_COMMENT_LIMIT + SB_QUAL_LIMIT) / (1024ULL * 1024ULL * 1024ULL);
        fprintf(stderr, "[M::%-25s] allocated %.2f GB on host for superbatch\n", __func__, nGB_allocated);
    }
    return batch;
}

/**
 * @brief remove data from a superbatch data set
 */
static void resetSuperBatchData(superbatch_data_t *data)
{
    data->n_reads = 0;
    data->name_size = 0;
    data->comment_size = 0;
    data->seqs_size = 0;
    data->qual_size = 0;
}

/**
 * @brief compare 2 reads a and b.
 * @return int positive if a > b, negative if a < b, 0 if a == b
 */
static int compareReads(const void *a, const void *b)
{
    char *a_key = ((bseq1_t *)a)->seq;
    char *b_key = ((bseq1_t *)b)->seq;
    return strncmp(a_key, b_key, 500);
}

/**
 * @brief sort reads lexicographically
 */
static void sortReads(bseq1_t *reads, int n_reads)
{
    qsort(reads, n_reads, sizeof(bseq1_t), compareReads);
}

/**
 * @brief
 *
 * @param ks
 * @param ks2
 * @param chunk_size
 * @param copy_comment
 * @param transfer_data
 * @return int number of reads loaded from file
 */
static unsigned long long dataloader(kseq_t *ks, kseq_t *ks2, unsigned long long loading_batch_size, int copy_comment, superbatch_data_t *transfer_data, g3_opt_t *g3_opt,
        double *func_elapsed_ms)
{
    FUNC_TIMER_START;
    int64_t size = 0;
    unsigned long long n_seqs_read;
    bseq_read2(loading_batch_size, &n_seqs_read, ks, ks2, transfer_data, g3_opt); // this will write to transfer_data
    bseq1_t *reads = transfer_data->reads;
    transfer_data->n_reads = n_seqs_read;
    if(n_seqs_read == 0){
        FUNC_TIMER_END;
        return 0;
    }
    if(copy_comment)
        for (int i = 0; i < n_seqs_read; ++i)
        {
            reads[i].comment = 0;
        }

    // sortReads(reads, n_seqs_read);
    FUNC_TIMER_END;
    return n_seqs_read;
}

/**
 * @brief process all data in fasta files using super batches
 *
 * @param aux top-level data on this program: input fasta files, indexes, mapping parameters.
 */
void main_gcube(ktp_aux_t *aux, g3_opt_t *g3_opt)
{
    struct timespec start, end;
    double walltime_initialize, walltime_process, walltime_cleanup;
    clock_gettime(CLOCK_REALTIME,  &start);

    int num_gpus;
    int num_use_gpus = g3_opt->num_use_gpus;
    hipGetDeviceCount(&num_gpus);
    if(num_gpus < num_use_gpus){
        std::cerr << "!! invalid request of " << num_use_gpus << " GPUs"
            << "where only " << num_gpus << " GPUs are available." << std::endl;
        exit(1);
    } else{
        std::cerr << "* using " << num_use_gpus << " GPUs out of " << num_gpus
            << " available GPUs." << std::endl;
    }

    // Double buffers to overlap processing and loading the next input batch.
    superbatch_data_t *loaded = newSuperBatchData();
    superbatch_data_t *loading = newSuperBatchData();

    // Double buffers per GPU to overlap processing and communicating,
    //   results of the previous batch and the next input batch (mini-batch).
    process_data_t *proc[MAX_NUM_GPUS];
    transfer_data_t *tran[MAX_NUM_GPUS];

    // Initialize host and device memory for processing with each GPU.
    //   Utilizes pinned & fixed memory for optimal I/O performance.
    for(int j=0; j<num_use_gpus; j++){
        newProcess(&proc[j], j, aux->opt, aux->pes0, aux->idx->bwt,\
                aux->idx->bns, aux->idx->pac, aux->kmerHashTab,\
                &(aux->loadedIndex), g3_opt);
        newTransfer(&tran[j], j, g3_opt);
        tran[j]->fd_outfile = aux->fd_outfile;
    }

    clock_gettime(CLOCK_REALTIME,  &end);
    walltime_initialize = (end.tv_sec - start.tv_sec) +\
                           (end.tv_nsec - start.tv_nsec) / 1e9;

    clock_gettime(CLOCK_REALTIME,  &start);
    long int num_total_reads = 0;
    memset(tprof, 0, sizeof(float) * MAX_NUM_GPUS * MAX_NUM_STEPS);


    // Load the next (global) input batch from the host storage
    //   and process the currently loaded batch using all GPUs.
    //   The offloader thread handles all communication and kernel invocation
    //   for GPUs. It also writes results to a single file in host storage.
#define A 0
#define B 1
#define toggle(ab) (1 - (ab))
    int AB = A;
    double load_elapsed_ms;
    double align_elapsed_ms;
    do { // TODO we want dataloader to be multi-threaded as well.
        std::thread t_dataloader(dataloader, aux->ks, aux->ks2, 
                aux->loading_batch_size, aux->copy_comment, loading, g3_opt,
                &load_elapsed_ms);
        std::thread t_offloader(offloader, loaded, proc, tran, g3_opt,
                &align_elapsed_ms);//, superbatch_results[AB]);
        //std::thread t_storer(storer, aux->fd_outfile, superbatch_results[toggle(AB)]); // possibly merge pulled results

        t_offloader.join();
        t_dataloader.join();

        fprintf(stderr, "* loaded %ld reads from storage in %.2f ms\n",
                loading->n_reads, load_elapsed_ms);
        fprintf(stderr, "* aligned %ld reads with %d GPUs in %.2f ms\n",
                loaded->n_reads, g3_opt->num_use_gpus, align_elapsed_ms);

        superbatch_data_t * tmp = loaded;
        loaded = loading;
        loading = tmp;
        resetSuperBatchData(loading);
        AB = toggle(AB);
        num_total_reads += loaded->n_reads;
    } while (loaded->n_reads != 0);

    clock_gettime(CLOCK_REALTIME,  &end);
    walltime_process = (end.tv_sec - start.tv_sec) +\
                           (end.tv_nsec - start.tv_nsec) / 1e9;



    // Destroy all generated per-GPU structures.
    for(int j=0; j<num_use_gpus; j++){
        hipStreamDestroy(*(hipStream_t*)proc[j]->CUDA_stream);
        hipStreamDestroy(*(hipStream_t*)tran[j]->CUDA_stream);
    }


    fprintf(stderr,"* Wall-clock time mem alloc & transfer: %.2lf seconds\n", walltime_initialize);
    fprintf(stderr,"* Wall-clock time for processing all %ld reads: %.2lf seconds\n", num_total_reads, walltime_process);


    std::cerr << std::endl << "* Wall-clock time for alignment across "
        << g3_opt->num_use_gpus << " GPUs for each stage (avg, min, max):"
        << std::endl;

    // Runtime profiling stats
    //  0: min, 1: avg, 2: max.
    float walltime_seeding[3], walltime_chaining[3], walltime_extending[3];
    //  0: seeding, 1: chaining, 2: extending.
    float tim, min_tim[3], max_tim[3], sum_tim[3];
    for(int k=0; k<3; k++){
        min_tim[k] = FLT_MAX; max_tim[k] = FLT_MIN; sum_tim[k] = 0;
    }
    float *tims;
    for(int gpuid = 0; gpuid < g3_opt->num_use_gpus; gpuid++){
        tims = tprof[gpuid];

        tim = tims[S_SMEM] + tims[S_R2] + tims[S_R3];
        if(tim < min_tim[0]) min_tim[0] = tim;
        if(tim > max_tim[0]) max_tim[0] = tim;
        sum_tim[0] += tim;

        tim = tims[C_SAL] + tims[C_SORT_SEEDS] + tims[C_CHAIN]
            + tims[C_SORT_CHAINS] + tims[C_FILTER];
        if(tim < min_tim[1]) min_tim[1] = tim;
        if(tim > max_tim[1]) max_tim[1] = tim;
        sum_tim[1] += tim;

        tim = tims[E_PAIRGEN] + tims[E_EXTEND] + tims[E_FILTER_MARK]
            + tims[E_SORT_ALNS] + tims[E_T_PAIRGEN] + tims[E_TRACEBACK]
            + tims[E_FINALIZE];
        if(tim < min_tim[2]) min_tim[2] = tim;
        if(tim > max_tim[2]) max_tim[2] = tim;
        sum_tim[2] += tim;
    }
    std::cerr << "\t\t\t\t\t- seeding: (";
    std::cerr << std::fixed << std::setprecision(2)
        << sum_tim[0] / g3_opt->num_use_gpus / 1000 << ", "
        << min_tim[0] / 1000 << ", "
        << max_tim[0] / 1000 << ") seconds" << std::endl;

    std::cerr << "\t\t\t\t\t- chaining: (";
    std::cerr << std::fixed << std::setprecision(2)
        << sum_tim[1] / g3_opt->num_use_gpus / 1000 << ", "
        << min_tim[1] / 1000 << ", "
        << max_tim[1] / 1000 << ") seconds" << std::endl;

    std::cerr << "\t\t\t\t\t- extending: (";
    std::cerr << std::fixed << std::setprecision(2)
        << sum_tim[2] / g3_opt->num_use_gpus / 1000 << ", "
        << min_tim[2] / 1000 << ", "
        << max_tim[2] / 1000 << ") seconds" << std::endl;


    for(int gpuid = 0; gpuid < g3_opt->num_use_gpus; gpuid++){
        std::cerr << std::endl
            << "* Wall-clock time, GPU #" << gpuid
            << " total alignment Sum: ";
        std::cerr << std::fixed << std::setprecision(2) 
            << tprof[gpuid][ALIGN_TOTAL] / 1000 << " seconds" << std::endl;
    }
}
