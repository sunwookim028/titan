#include "bntseq_CUDA.cuh"
#include "CUDAKernel_memmgnt.cuh"

#define _set_pac(pac, l, c) ((pac)[(l)>>2] |= (c)<<((~(l)&3)<<1))
#define _get_pac(pac, l) ((pac)[(l)>>2]>>((~(l)&3)<<1)&3)

__device__ int bns_pos2rid_gpu(const bntseq_t *bns, int64_t pos_f)
{
	int left, mid, right;
	if (pos_f >= bns->l_pac) return -1;
	left = 0; mid = 0; right = bns->n_seqs;
	while (left < right) { // binary search
		mid = (left + right) >> 1;
		if (pos_f >= bns->anns[mid].offset) {
			if (mid == bns->n_seqs - 1) break;
			if (pos_f < bns->anns[mid+1].offset) break; // bracketed
			left = mid + 1;
		} else right = mid;
	}
	return mid;
}

__device__ static inline int64_t bns_depos_gpu(const bntseq_t *bns, int64_t pos, int *is_rev)
{
	return (*is_rev = (pos >= bns->l_pac))? (bns->l_pac<<1) - 1 - pos : pos;
}


__device__ int bns_intv2rid_gpu(const bntseq_t *bns, int64_t rb, int64_t re)
{
	int is_rev, rid_b, rid_e;
	if (rb < bns->l_pac && re > bns->l_pac) return -2;
	rid_b = bns_pos2rid_gpu(bns, bns_depos_gpu(bns, rb, &is_rev));
	rid_e = rb < re? bns_pos2rid_gpu(bns, bns_depos_gpu(bns, re - 1, &is_rev)) : rid_b;
	return rid_b == rid_e? rid_b : -1;
}


__device__ uint8_t *bns_get_seq_gpu(int64_t l_pac, const uint8_t *pac, int64_t beg, int64_t end, int64_t *len, void* d_buffer_ptr)
{
	uint8_t *seq = 0;
	if (end < beg) end ^= beg, beg ^= end, end ^= beg; // if end is smaller, swap
	if (end > l_pac<<1) end = l_pac<<1;
	if (beg < 0) beg = 0;
	if (beg >= l_pac || end <= l_pac) {
		int64_t k, l = 0;
		*len = end - beg;
		seq = (uint8_t*)CUDAKernelMalloc(d_buffer_ptr, end - beg, 1);
		if (beg >= l_pac) { // reverse strand
			int64_t beg_f = (l_pac<<1) - 1 - end;
			int64_t end_f = (l_pac<<1) - 1 - beg;
			for (k = end_f; k > beg_f; --k)
				seq[l++] = 3 - _get_pac(pac, k);
		} else { // forward strand
			for (k = beg; k < end; ++k)
				seq[l++] = _get_pac(pac, k);
		}
	} else *len = 0; // if bridging the forward-reverse boundary, return nothing
	return seq;
}

__device__ uint8_t *bns_fetch_seq_gpu(const bntseq_t *bns, const uint8_t *pac, int64_t *beg, int64_t mid, int64_t *end, int *rid, void* d_buffer_ptr)
{
	int64_t far_beg, far_end, len;
	int is_rev;
	uint8_t *seq;

	if (*end < *beg) *end ^= *beg, *beg ^= *end, *end ^= *beg; // if end is smaller, swap
	// assert(*beg <= mid && mid < *end);
	*rid = bns_pos2rid_gpu(bns, bns_depos_gpu(bns, mid, &is_rev));
	far_beg = bns->anns[*rid].offset;
	far_end = far_beg + bns->anns[*rid].len;
	if (is_rev) { // flip to the reverse strand
		int64_t tmp = far_beg;
		far_beg = (bns->l_pac<<1) - far_end;
		far_end = (bns->l_pac<<1) - tmp;
	}
	*beg = *beg > far_beg? *beg : far_beg;
	*end = *end < far_end? *end : far_end;
	seq = bns_get_seq_gpu(bns->l_pac, pac, *beg, *end, &len, d_buffer_ptr);
	// if (seq == 0 || *end - *beg != len) {
	// 	fprintf(stderr, "[E::%s] begin=%ld, mid=%ld, end=%ld, len=%ld, seq=%p, rid=%d, far_beg=%ld, far_end=%ld\n",
	// 			__func__, (long)*beg, (long)mid, (long)*end, (long)len, seq, *rid, (long)far_beg, (long)far_end);
	// }
	// assert(seq && *end - *beg == len); // assertion failure should never happen
	return seq;
}