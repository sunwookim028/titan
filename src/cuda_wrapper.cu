#include "bwa.h"
#include "cuda_wrapper.h"
#include "gmem_alloc.h"
#include "macro.h"
#include "timer.h"
#include <iostream>


/* transfer index data */
static void transferIndex(
	const bwt_t *bwt, 
	const bntseq_t *bns, 
	const uint8_t *pac,
	const kmers_bucket_t *kmerHashTab,
	process_data_t *process_instance,
    unsigned long long *allocated_size)
{
		/* CUDA GLOBAL MEMORY ALLOCATION AND TRANSFER */

	// Burrows-Wheeler Transform
		// 1. bwt_t structure
	//if (bwa_verbose>=3) fprintf(stderr, "[M::%-25s] *** bwt .......... %.2f MB\n", __func__, (float)sizeof(bwt_t)/MB_SIZE);
	bwt_t* d_bwt;
	hipMalloc((void**)&d_bwt, sizeof(bwt_t));
	hipMemcpy(d_bwt, bwt, sizeof(bwt_t), hipMemcpyHostToDevice);
		// 2. int array of bwt
	//if (bwa_verbose>=3) fprintf(stderr, "[M::%-25s] *** bwt_int ...... %.2f MB\n", __func__, (float)bwt->bwt_size*sizeof(uint32_t)/MB_SIZE);
	uint32_t* d_bwt_int ;
	hipMalloc((void**)&d_bwt_int, bwt->bwt_size*sizeof(uint32_t));
	hipMemcpy(d_bwt_int, bwt->bwt, bwt->bwt_size*sizeof(uint32_t), hipMemcpyHostToDevice);
		// 3. int array of Suffix Array
	//if (bwa_verbose>=3) fprintf(stderr, "[M::%-25s] *** suffix array . %.2f MB \n", __func__, (float)bwt->n_sa*sizeof(bwtint_t)/MB_SIZE);
	bwtint_t* d_bwt_sa ;
	hipMalloc((void**)&d_bwt_sa, bwt->n_sa*sizeof(bwtint_t));
	hipMemcpy(d_bwt_sa, bwt->sa, bwt->n_sa*sizeof(bwtint_t), hipMemcpyHostToDevice);
		// set pointers on device's memory to bwt_int and SA
	hipMemcpy((void**)&(d_bwt->bwt), &d_bwt_int, sizeof(uint32_t*), hipMemcpyHostToDevice);
	hipMemcpy((void**)&(d_bwt->sa), &d_bwt_sa, sizeof(bwtint_t*), hipMemcpyHostToDevice);


	unsigned long long total_size = sizeof(bwt_t) +\
                                    bwt->bwt_size*sizeof(uint32_t) +\
                                    bwt->n_sa*sizeof(bwtint_t) +\
                                    bns->n_seqs*sizeof(bntann1_t) +\
                                    bns->n_holes*sizeof(bntamb1_t) +\
                                    bns->l_pac*sizeof(uint8_t);
	//fprintf(stderr, "[M::%-25s] Device memory for Index ...... %.2f MB \n", __func__, (float)total_size/MB_SIZE);

	// BNS
	// First create h_bns as a copy of bns on host
	// Then allocate its member pointers on device and copy data over
	// Then copy h_bns to d_bns
	uint32_t i, size;			// loop index and length of strings
	bntseq_t* h_bns;			// host copy to modify pointers
	h_bns = (bntseq_t*)malloc(sizeof(bntseq_t));
	memcpy(h_bns, bns, sizeof(bntseq_t));
	h_bns->anns = (bntann1_t*)malloc(bns->n_seqs*sizeof(bntann1_t));
	memcpy(h_bns->ambs, bns->ambs, bns->n_holes*sizeof(bntamb1_t));
	h_bns->ambs = (bntamb1_t*)malloc(bns->n_holes*sizeof(bntamb1_t));
	memcpy(h_bns->anns, bns->anns, bns->n_seqs*sizeof(bntann1_t));

		// allocate anns.name
	for (i=0; i<bns->n_seqs; i++){
		size = strlen(bns->anns[i].name);
		// allocate this name and copy to device
		hipMalloc((void**)&(h_bns->anns[i].name), size+1); 			// +1 for "\0"
		hipMemcpy(h_bns->anns[i].name, bns->anns[i].name, size+1, hipMemcpyHostToDevice);
	}
	// allocate anns.anno
	for (i=0; i<bns->n_seqs; i++){
		size = strlen(bns->anns[i].anno);
		// allocate this name and copy to device
		hipMalloc((void**)&(h_bns->anns[i].anno), size+1); 			// +1 for "\0"
		hipMemcpy(h_bns->anns[i].anno, bns->anns[i].anno, size+1, hipMemcpyHostToDevice);
	}
		// now h_bns->anns has pointers of name and anno on device
		// allocate anns on device and copy data from h_bns->anns to device
	bntann1_t* temp_d_anns;
	//if (bwa_verbose>=3) fprintf(stderr, "[M::%-25s] *** bns.anns ..... %.2f MB\n", __func__, (float)bns->n_seqs*sizeof(bntann1_t)/MB_SIZE);
	hipMalloc((void**)&temp_d_anns, bns->n_seqs*sizeof(bntann1_t));
	hipMemcpy(temp_d_anns, h_bns->anns, bns->n_seqs*sizeof(bntann1_t), hipMemcpyHostToDevice);
		// now assign this pointer to h_bns->anns
	h_bns->anns = temp_d_anns;

		// allocate bns->ambs on device and copy data to device
	//if (bwa_verbose>=3) fprintf(stderr, "[M::%-25s] *** bns.ambs ..... %.2f MB\n", __func__, (float)bns->n_holes*sizeof(bntamb1_t)/MB_SIZE);
	hipMalloc((void**)&h_bns->ambs, bns->n_holes*sizeof(bntamb1_t));
	hipMemcpy(h_bns->ambs, bns->ambs, bns->n_holes*sizeof(bntamb1_t), hipMemcpyHostToDevice);

		// finally allocate d_bns and copy from h_bns
	//if (bwa_verbose>=3) fprintf(stderr, "[M::%-25s] *** bns .......... %.2f MB\n", __func__, (float)sizeof(bntseq_t)/MB_SIZE);
	bntseq_t* d_bns;
	hipMalloc((void**)&d_bns, sizeof(bntseq_t));
	hipMemcpy(d_bns, h_bns, sizeof(bntseq_t), hipMemcpyHostToDevice);

	// PAC
	//if (bwa_verbose>=3) fprintf(stderr, "[M::%-25s] *** pac .......... %.2f MB\n", __func__, (float)bns->l_pac*sizeof(uint8_t)/MB_SIZE);
	uint8_t* d_pac ;
	hipMalloc((void**)&d_pac, bns->l_pac/4*sizeof(uint8_t)); 		// l_pac is length of ref seq
	hipMemcpy(d_pac, pac, bns->l_pac/4*sizeof(uint8_t), hipMemcpyHostToDevice); 		// divide by 4 because 2-bit encoding

	// K-MER HASH TABLE
	//if (bwa_verbose>=3) fprintf(stderr, "[M::%-25s] *** kmer ......... %.2f MB\n", __func__, (float)pow4(KMER_K)*sizeof(kmers_bucket_t)/MB_SIZE);
	kmers_bucket_t* d_kmerHashTab ;
	hipMalloc((void**)&d_kmerHashTab, pow4(KMER_K)*sizeof(kmers_bucket_t)); 		// l_pac is length of ref seq
	hipMemcpy(d_kmerHashTab, kmerHashTab, pow4(KMER_K)*sizeof(kmers_bucket_t), hipMemcpyHostToDevice); 		// divide by 4 because 2-bit encoding


	// output
	process_instance->d_bwt = d_bwt;
	process_instance->d_bns = d_bns;
	process_instance->d_pac = d_pac;
	process_instance->d_kmerHashTab = d_kmerHashTab;
    std::cerr << "* bwt index " << total_size / MB_SIZE << " MB\n";
}

/* transfer user-defined optinos */
static void transferOptions(
	const mem_opt_t *opt, 
	mem_pestat_t *pes0,
	process_data_t *process_instance,
    unsigned long long *allocated_size)
{
	// matching and mapping options (opt)
	mem_opt_t* d_opt;
	hipMalloc((void**)&d_opt, sizeof(mem_opt_t));
	hipMemcpy(d_opt, opt, sizeof(mem_opt_t), hipMemcpyHostToDevice);

	// paired-end stats: only allocate on device
	mem_pestat_t* d_pes;
	if (opt->flag&MEM_F_PE){
		//fprintf(stderr, "[M::%-25s] pestat ....... %.2f MB\n", __func__, (float)4*sizeof(mem_pestat_t)/MB_SIZE);
		hipMalloc((void**)&d_pes, 4*sizeof(mem_pestat_t));
	}

	// output
	process_instance->d_opt = d_opt;
	process_instance->d_pes = d_pes;
	process_instance->h_pes0 = pes0;
}

/* transfer index data */
static void transferFmIndex(
        process_data_t *process_instance,
        const fmIndex *idx,
    unsigned long long *allocated_size)
{
    /**
     * Reference data to transfer:
     *      count, count2
     *      cpOcc, cpOcc2
     *      oneHot, sentinelIndex, firstBase
     */
    fmIndex hostFmIndex;
    hipError_t err;

    long long size = 0;

    // FOR BWT-2
    uint64_t *d_one_hot;
    int sizeOneHot = 64 * sizeof(uint64_t);
    size += sizeOneHot;
    CUDA_CHECK(hipMalloc((void**)&d_one_hot, sizeOneHot));
    CUDA_CHECK(hipMemcpy(d_one_hot, idx->oneHot, sizeOneHot, hipMemcpyHostToDevice));

    CP_OCC *d_cp_occ;
    int64_t cp_occ_size = idx->cpOccSize;
    size += cp_occ_size*sizeof(CP_OCC);
    CUDA_CHECK(hipMalloc((void**)&d_cp_occ, cp_occ_size*sizeof(CP_OCC)));
    CUDA_CHECK(hipMemcpy(d_cp_occ, idx->cpOcc, cp_occ_size*sizeof(CP_OCC), hipMemcpyHostToDevice));

    int64_t *d_count;
    int sizeCount = sizeof(int64_t) * 5;
    size += sizeCount;
    CUDA_CHECK(hipMalloc((void**)&d_count, sizeCount));
    CUDA_CHECK(hipMemcpy(d_count, idx->count, sizeCount, hipMemcpyHostToDevice));


    CP_OCC2 *d_cp_occ2;
    size += cp_occ_size*sizeof(CP_OCC2);
    CUDA_CHECK(hipMalloc((void**)&d_cp_occ2, cp_occ_size*sizeof(CP_OCC2)));
    CUDA_CHECK(hipMemcpy(d_cp_occ2, idx->cpOcc2, cp_occ_size*sizeof(CP_OCC2), hipMemcpyHostToDevice));

    int64_t *d_count2;
    int sizeCount2 = sizeof(int64_t) * 17;
    size += sizeCount2;
    CUDA_CHECK(hipMalloc((void**)&d_count2, sizeCount2));
    CUDA_CHECK(hipMemcpy(d_count2, idx->count2, sizeCount2, hipMemcpyHostToDevice));

    uint8_t *d_first_base;
    CUDA_CHECK(hipMalloc((void**)&d_first_base, sizeof(uint8_t)));
    CUDA_CHECK(hipMemcpy(d_first_base, idx->firstBase, sizeof(uint8_t), hipMemcpyHostToDevice));

    int64_t *deviceSentinelIndex;
    CUDA_CHECK(hipMalloc((void**)&deviceSentinelIndex, sizeof(int64_t)));
    CUDA_CHECK(hipMemcpy(deviceSentinelIndex, idx->sentinelIndex, sizeof(int64_t), hipMemcpyHostToDevice));

    hostFmIndex.oneHot = d_one_hot;
    hostFmIndex.cpOcc = d_cp_occ;
    hostFmIndex.cpOcc2 = d_cp_occ2;
    hostFmIndex.count = d_count;
    hostFmIndex.count2 = d_count2;
    hostFmIndex.firstBase = d_first_base;
    hostFmIndex.sentinelIndex = deviceSentinelIndex;

    fmIndex *deviceFmIndex;
    CUDA_CHECK(hipMalloc((void**)&deviceFmIndex, sizeof(fmIndex)));
    CUDA_CHECK(hipMemcpy(deviceFmIndex, &hostFmIndex, sizeof(fmIndex), hipMemcpyHostToDevice));

    std::cerr << "* occ2 index: " << size / MB_SIZE << " MB\n";

    // output
    process_instance->d_fmIndex = deviceFmIndex;
}




process_data_t * device_alloc(
        int gpuid,
        pipeline_aux_t *aux
        )
{
    CUDA_CHECK(hipSetDevice(gpuid));
    int current;
    CUDA_CHECK(hipGetDevice(&current));
    if(current != gpuid){
        exit(1);
    }
    process_data_t *proc = new process_data_t;
    proc->gpu_no = gpuid;

    unsigned long long size;

	// dynamic allocation pool management
	proc->d_buffer_pools = CUDA_BufferInit(CUDA_MALLOC_CAP);
    std::cerr << "* device " << gpuid << " allocating " 
        << CUDA_MALLOC_CAP / MB_SIZE << " MB for dynamic allocation pool\n";

	// initialize intermediate processing memory on device
    size = (sizeof(smem_aux_t) + sizeof(mem_seed_v) + sizeof(mem_chain_v)
            + sizeof(seed_record_t) * MAX_NUM_SW_SEEDS + sizeof(mem_alnreg_v)
            + sizeof(mem_aln_v) + 20 * sizeof(int)) * MAX_BATCH_SIZE 
            + sizeof(int);
    CUDA_CHECK(hipMalloc(&proc->d_aux, sizeof(smem_aux_t) * MAX_BATCH_SIZE));
	CUDA_CHECK(hipMalloc(&proc->d_seq_seeds, MAX_BATCH_SIZE *sizeof(mem_seed_v)));
	CUDA_CHECK(hipMalloc(&proc->d_chains, MAX_BATCH_SIZE *sizeof(mem_chain_v)));
	CUDA_CHECK(hipMalloc(&proc->d_seed_records, MAX_BATCH_SIZE *MAX_NUM_SW_SEEDS*sizeof(seed_record_t)));
	CUDA_CHECK(hipMalloc(&proc->d_Nseeds, sizeof(int)));
	CUDA_CHECK(hipMalloc(&proc->d_regs, MAX_BATCH_SIZE *sizeof(mem_alnreg_v)));
	CUDA_CHECK(hipMalloc(&proc->d_alns, MAX_BATCH_SIZE *sizeof(mem_aln_v)));

	CUDA_CHECK(hipMalloc(&proc->d_sortkeys_in, MAX_BATCH_SIZE *5*sizeof(int)));
	CUDA_CHECK(hipMalloc(&proc->d_sortkeys_out, MAX_BATCH_SIZE *5*sizeof(int)));
	CUDA_CHECK(hipMalloc(&proc->d_seqIDs_in, MAX_BATCH_SIZE *5*sizeof(int)));
	CUDA_CHECK(hipMalloc(&proc->d_seqIDs_out, MAX_BATCH_SIZE *5*sizeof(int)));
    if(proc->d_aux && proc->d_seq_seeds && proc->d_chains 
            && proc->d_seed_records && proc->d_Nseeds && proc->d_regs
            && proc->d_alns && proc->d_sortkeys_in && proc->d_sortkeys_out
            && proc->d_seqIDs_in && proc->d_seqIDs_out){
        std::cerr << "* device " << gpuid 
            << " intermediate data " << size / MB_SIZE << " MB\n";
    } else{
        std::cerr << "* device " << gpuid 
            << " intermediate data alloc failed\n";
        exit(EXIT_FAILURE);
    }


    // input on device
    size = MAX_BATCH_SIZE * sizeof(uint8_t) * MAX_LEN_READ;
    CUDA_CHECK(hipMalloc(&proc->d_seq, size));
    std::cerr << "* device " << gpuid << " allocating " 
        << size / MB_SIZE << " MB for input seqs\n";
    if(proc->d_seq == nullptr){
        std::cerr << "Error.  device memory for copying seqs.\n";
        exit(EXIT_FAILURE);
    }
    size = sizeof(int) * (MAX_BATCH_SIZE + 1);
    CUDA_CHECK(hipMalloc(&proc->d_seq_offset, size));
    std::cerr << "* device " << gpuid << " allocating " 
        << size / MB_SIZE << " MB for input seq offsets\n";
	if(proc->d_seq_offset == nullptr){
        std::cerr << "Error.  device memory for copying offsets.\n";
        exit(EXIT_FAILURE);
    }


    CUDA_CHECK(hipMalloc(&proc->d_alns_offset, 
                sizeof(int) * (MAX_BATCH_SIZE + 1)));
    CUDA_CHECK(hipMalloc(&proc->d_rid,
                sizeof(int) * (MAX_ALN_CNT + 1)));
    CUDA_CHECK(hipMalloc(&proc->d_pos,
                sizeof(uint64_t) * (MAX_ALN_CNT + 1)));
    CUDA_CHECK(hipMalloc(&proc->d_chunk_aln_count,
                sizeof(int)));
    if(proc->d_alns_offset == nullptr || 
            proc->d_rid == nullptr ||
            proc->d_pos == nullptr ||
            proc->d_chunk_aln_count == nullptr){
        std::cerr << "hipMalloc err.\n";
        exit(EXIT_FAILURE);
    }


    /*
	// pinned memory for memcpy
    size = g3_opt->batch_size * sizeof(uint8_t) * MAX_LEN_READ;
    CUDA_CHECK(hipHostMalloc(&proc->h_seq, size));
    std::cerr << "* allocating " 
        << size << " bytes for input seq pinned memcpy\n";
	if(proc->h_seq == nullptr){
        std::cerr << "Error. Host pinned memory for copying seqs.\n";
        exit(EXIT_FAILURE);
    }
    size = (g3_opt->batch_size + 1) * sizeof(int);
    CUDA_CHECK(hipHostMalloc(&proc->h_seq_offset, size));
    std::cerr << "* allocating " 
        << size << " bytes for input seq offsets pinned memcpy\n";
	if(proc->h_seq_offset == nullptr){
        std::cerr << "Error. Host pinned memory for copying offsets.\n";
        exit(EXIT_FAILURE);
    }
    */

	// initialize a cuda stream for processing
	//proc->CUDA_stream = malloc(sizeof(hipStream_t));
	//CUDA_CHECK(hipStreamCreate((hipStream_t*)proc->CUDA_stream));

    return proc;
}

void memcpy_index(
        process_data_t *instance,
        int gpuid, 
        pipeline_aux_t *aux
        )
{
    TIMER_INIT();
    TIMER_START();
    CUDA_CHECK(hipSetDevice(gpuid));
    int current;
    CUDA_CHECK(hipGetDevice(&current));
    if(current != gpuid){
        std::cerr << "GPU " << gpuid << "  " << "* device_alloc: hipSetDevice is wrong" << std::endl;
        exit(1);
    }
    unsigned long long size;

	// user-defined options
	transferOptions(aux->opt, aux->pes0, instance, &size);
    
	// transfer index data
	transferIndex(aux->idx->bwt, aux->idx->bns, aux->idx->pac, 
            aux->kmerHashTab, instance, &size);

    transferFmIndex(instance, &(aux->loadedIndex), &size);
    TIMER_END(0, "");
    tprof[gpuid][GPU_SETUP] = duration.count() / 1000;
}


// no pinned memcpy for now.
void memcpy_input(int batch_size, process_data_t *proc,
        uint8_t *seq, int *seq_offset)
{
    int total_seq_len = seq_offset[batch_size];
    size_t size;
    size = sizeof(uint8_t) * total_seq_len;
    CUDA_CHECK(hipMemcpy(proc->d_seq, seq, size, hipMemcpyHostToDevice));
    size = sizeof(int) * (batch_size + 1);
    CUDA_CHECK(hipMemcpy(proc->d_seq_offset, seq_offset, size,
            hipMemcpyHostToDevice));
    proc->batch_size = batch_size;

    // reset
    int zero_int = 0;
    CUDA_CHECK(hipMemcpy(proc->d_Nseeds, &zero_int, sizeof(int), hipMemcpyHostToDevice));
    CUDAResetBufferPool(proc->d_buffer_pools);
}




void check_device_count(int num_requested_gpus)
{
    int num_available_gpus;
    CUDA_CHECK(hipGetDeviceCount(&num_available_gpus));
    if(num_available_gpus < num_requested_gpus){
        std::cerr << "!! invalid request of " << num_requested_gpus 
            << " GPUs where only " << num_available_gpus << " GPUs are available.";
        exit(1);
    } else{
        std::cerr << "* using " << num_requested_gpus << 
            " GPUs out of " << num_available_gpus << " available GPUs.\n";
    }
}

void destruct_proc(process_data_t *proc)
{
    //hipStreamDestroy(*(hipStream_t*)proc->CUDA_stream);
}

void cuda_wrapper_test()
{
    std::chrono::high_resolution_clock::time_point start, end;
    std::chrono::duration<long long, std::micro> duration;
    start = std::chrono::high_resolution_clock::now();

    hipFree(0);

    end = std::chrono::high_resolution_clock::now();
    duration = std::chrono::duration_cast<std::chrono::microseconds>(end - start);
    std::cerr << "* cuda init: " << duration.count() / 1000 << " ms" << std::endl;
}


void memcpy_output(
        aligned_chunk *ac,
        process_data_t * proc)
{
    int num_alns;

    ac->aln_offsets.resize(ac->chunk_size + 1);
    CUDA_CHECK(hipMemcpy(ac->aln_offsets.data(),
                proc->d_alns_offset,
                sizeof(int) * ac->chunk_size + 1,
                hipMemcpyDeviceToHost));
    
    // somehow this is needed, since ac->aln_offsets[ac->chunk_size]
    // appears to contain a different, corrupted value. 
    CUDA_CHECK(hipMemcpy(&num_alns,
                proc->d_alns_offset + ac->chunk_size,
                sizeof(int),
                hipMemcpyDeviceToHost));

    ac->rid.resize(num_alns);
    ac->pos.resize(num_alns);
    CUDA_CHECK(hipMemcpy(ac->rid.data(),
                proc->d_rid,
                sizeof(int) * num_alns,
                hipMemcpyDeviceToHost));
    CUDA_CHECK(hipMemcpy(ac->pos.data(),
                proc->d_pos,
                sizeof(uint64_t) * num_alns,
                hipMemcpyDeviceToHost));
}


#if 0

/* copy sam output to host */
void CUDATransferSamOut(transfer_data_t *transfer_data){
    int gpuid = transfer_data->gpu_no;
    if(hipSetDevice(transfer_data->gpu_no) != hipSuccess){
        std::cerr << "CUDATransferSamOut: hipSetDevice failed" << std::endl;
        exit(1);
    }
    int current;
    hipGetDevice(&current);
    if(current != gpuid){
         std::cerr << "GPU " << gpuid << "  " << "CUDATransferSamOut: hipSetDevice is wrong" << std::endl;
        return;
    }
	hipStream_t *transfer_stream = (hipStream_t*)(transfer_data->CUDA_stream);
	CUDA_CHECK(hipMemcpyAsync(transfer_data->h_seqs, transfer_data->d_seqs, transfer_data->batch_size*sizeof(bseq1_t), hipMemcpyDeviceToHost, *transfer_stream));
	// transfer all SAM from device to host
	// first find the total size of all SAM's
	int sam_size;
	CUDA_CHECK(hipMemcpyAsync(&sam_size, transfer_data->d_seq_sam_size, sizeof(int), hipMemcpyDeviceToHost, *transfer_stream));
	// now copy
	CUDA_CHECK(hipMemcpyAsync(transfer_data->h_seq_sam_ptr, transfer_data->d_seq_sam_ptr, sam_size, hipMemcpyDeviceToHost, *transfer_stream));

	hipStreamSynchronize(*transfer_stream);

	// after GPU processing, seqs[i].sam are offset. Now we need to convert this offset to actual location
	bseq1_t *seqs = transfer_data->h_seqs;
	char *sam = transfer_data->h_seq_sam_ptr;
	for (int i=0; i<transfer_data->batch_size; i++)
		seqs[i].sam = sam + (long)seqs[i].sam;
}
#endif
