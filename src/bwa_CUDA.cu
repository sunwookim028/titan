#include "hip/hip_runtime.h"
#include "bwa_CUDA.cuh"
#include "bntseq_CUDA.cuh"
#include "CUDAKernel_memmgnt.cuh"
#include "ksw_CUDA.cuh"
#include "kstring_CUDA.cuh"

// Generate CIGAR when the alignment end points are known
__device__ uint32_t *bwa_gen_cigar2_gpu(const int8_t mat[25], int o_del, int e_del, int o_ins, int e_ins, int w_, int64_t l_pac, const uint8_t *pac, int l_query, uint8_t *query, int64_t rb, int64_t re, int *score, int *n_cigar, int *NM, void* d_buffer_ptr)
{
	uint32_t *cigar = 0;
	uint8_t tmp, *rseq;
	int i;
	int64_t rlen;
	kstring_t str;
	const char *int2base;

	if (n_cigar) *n_cigar = 0;
	if (NM) *NM = -1;
	if (l_query <= 0 || rb >= re || (rb < l_pac && re > l_pac)) return 0; // reject if negative length or bridging the forward and reverse strand
	rseq = bns_get_seq_gpu(l_pac, pac, rb, re, &rlen, d_buffer_ptr);
	if (re - rb != rlen) goto ret_gen_cigar; // possible if out of range
	if (rb >= l_pac) { // then reverse both query and rseq; this is to ensure indels to be placed at the leftmost position
		for (i = 0; i < l_query>>1; ++i)
			tmp = query[i], query[i] = query[l_query - 1 - i], query[l_query - 1 - i] = tmp;
		for (i = 0; i < rlen>>1; ++i)
			tmp = rseq[i], rseq[i] = rseq[rlen - 1 - i], rseq[rlen - 1 - i] = tmp;
	}
	if (l_query == re - rb && w_ == 0) { // no gap; no need to do DP
		// UPDATE: we come to this block now... FIXME: due to an issue in mem_reg2aln(), we never come to this block. This does not affect accuracy, but it hurts performance.
		if (n_cigar) {
			cigar = (uint32_t*)CUDAKernelMalloc(d_buffer_ptr, 4, 4);
			cigar[0] = l_query<<4 | 0;
			*n_cigar = 1;
		}
		for (i = 0, *score = 0; i < l_query; ++i)
			*score += mat[rseq[i]*5 + query[i]];
	} else {
		int w, max_gap, max_ins, max_del, min_w;
		// set the band-width
		max_ins = (int)((double)(((l_query+1)>>1) * mat[0] - o_ins) / e_ins + 1.);
		max_del = (int)((double)(((l_query+1)>>1) * mat[0] - o_del) / e_del + 1.);
		max_gap = max_ins > max_del? max_ins : max_del;
		max_gap = max_gap > 1? max_gap : 1;
		w = (max_gap + abs((int)rlen - l_query) + 1) >> 1;
		w = w < w_? w : w_;
		min_w = abs((int)rlen - l_query) + 3;
		w = w > min_w? w : min_w;
		// NW alignment
		*score = ksw_global2(l_query, query, rlen, rseq, 5, mat, o_del, e_del, o_ins, e_ins, w, n_cigar, &cigar, d_buffer_ptr);
	}
	if (NM && n_cigar) {// compute NM and MD
		int k, x, y, u, n_mm = 0, n_gap = 0;
		str.l = str.m = *n_cigar * 4; str.s = (char*)cigar; // append MD to CIGAR
		int2base = rb < l_pac? "ACGTN" : "TGCAN";
		for (k = 0, x = y = u = 0; k < *n_cigar; ++k) {
			int op, len;
			cigar = (uint32_t*)str.s;
			op  = cigar[k]&0xf, len = cigar[k]>>4;
			if (op == 0) { // match
				for (i = 0; i < len; ++i) {
					if (query[x + i] != rseq[y + i]) {
						kputw(u, &str, d_buffer_ptr);
						kputc(int2base[rseq[y+i]], &str, d_buffer_ptr);
						++n_mm; u = 0;
					} else ++u;
				}
				x += len; y += len;
			} else if (op == 2) { // deletion
				if (k > 0 && k < *n_cigar - 1) { // don't do the following if D is the first or the last CIGAR
					kputw(u, &str, d_buffer_ptr); kputc('^', &str, d_buffer_ptr);
					for (i = 0; i < len; ++i)
						kputc(int2base[rseq[y+i]], &str, d_buffer_ptr);
					u = 0; n_gap += len;
				}
				y += len;
			} else if (op == 1) x += len, n_gap += len; // insertion
		}
		kputw(u, &str, d_buffer_ptr); kputc(0, &str, d_buffer_ptr);
		*NM = n_mm + n_gap;
		cigar = (uint32_t*)str.s;
	}
	if (rb >= l_pac) // reverse back query
		for (i = 0; i < l_query>>1; ++i)
			tmp = query[i], query[i] = query[l_query - 1 - i], query[l_query - 1 - i] = tmp;

ret_gen_cigar:
	// free(rseq);
	return cigar;
}
