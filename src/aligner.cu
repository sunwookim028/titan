#include "timer.h"
#include "macro.h"
#include "bwa.h"
#include <locale.h>
#include "bwamem_GPU.cuh"
#include "streams.cuh"
#include <future>
#include <iostream>
#include <iomanip>
#include <atomic>
#include <mutex>
#include <sys/time.h>
#include <stdlib.h>
#include <string.h>
#include <math.h>
#include <unistd.h>
#include <iostream>
#include <ostream>
#include <thread>

extern float tprof[MAX_NUM_GPUS][MAX_NUM_STEPS];

/**
 * @brief convert current host addresses on a minibatch's transfer_data to their (future) addresses on GPU
 * assuming name, seq, comment, qual pointers on trasnfer_data still points to host memory
 *
 * @param seqs
 * @param n_seqs
 * @param transfer_data transfer_data_t object where these reads reside
 */
void convert2DevAddr(transfer_data_t *transfer_data)
{
	auto reads = transfer_data->h_seqs;
	auto n_reads = transfer_data->n_seqs;
	auto first_read = reads[0];
	for (int i = 0; i < n_reads; i++)
	{
		reads[i].name = reads[i].name - first_read.name + transfer_data->d_seq_name_ptr;
		reads[i].seq = reads[i].seq - first_read.seq + transfer_data->d_seq_seq_ptr;
		reads[i].comment = reads[i].comment - first_read.comment + transfer_data->d_seq_comment_ptr;
		reads[i].qual = reads[i].qual - first_read.qual + transfer_data->d_seq_qual_ptr;
	}
}

/**
 * @brief copy a minibatch of n_reads from superbatch to transfer_data minibatch's pinned memory, starting from firstReadId. 
 * Read info are contiguous, but name, comment, seq, qual are not
 * 
 * @param superbatch_data
 * @param transfer_data 
 * @param firstReadId 
 * @param n_reads 
 */
void copyReads2PinnedMem(superbatch_data_t *superbatch_data, transfer_data_t *transfer_data, int firstReadId, int n_reads){
	int lastReadId = firstReadId + n_reads - 1; 
	// copy name, comment, seq, qual one by one
	for (int i = firstReadId; i <= lastReadId; i++){
		bseq1_t *read = &(superbatch_data->reads[i]);
		char *toAddr;

		toAddr = transfer_data->h_seq_seq_ptr + transfer_data->h_seq_seq_size;
		memcpy(toAddr, read->seq, read->l_seq + 1); // size + 1 for null-terminating char
		read->seq = toAddr;
		transfer_data->h_seq_seq_size += read->l_seq + 1;
	}
	// copy read info
	memcpy(transfer_data->h_seqs, &superbatch_data->reads[firstReadId], n_reads * sizeof(bseq1_t));
}

/**
 * @brief load a small batch from superbatch to transfer_data, up to MB_MAX_COUNT. 
 * Return number of reads loaded into transfer_data->n_seqs. return 0 if no read loaded
 * after loading, translate reads' addresses to GPU and transfer to GPU,
 * @param transfer_data
 * @param superbatch_data
 * @param num_loaded number of reads loaded from this superbatch before this minibatch
 * @return int number of reads loaded into transfer_data->n_seqs
 */
static void push(transfer_data_t *tran, superbatch_data_t *loaded_input,
        int *push_counter, std::mutex *push_m, g3_opt_t *g3_opt,
        int *actual_push_count,
        double *func_elapsed_ms)
{
    FUNC_TIMER_START;
    int push_count;
    int push_offset;

    push_m->lock();
    push_offset = *push_counter;
    tran->batch_offset = loaded_input->loading_batch_offset + push_offset;

    if(push_offset >= loaded_input->n_reads){
        push_count = 0;
    } else{
        if(loaded_input->n_reads < g3_opt->batch_size){
            push_count = loaded_input->n_reads;
        } else{
            if(push_offset <= loaded_input->n_reads - g3_opt->batch_size){
                push_count = g3_opt->batch_size;
            } else if(push_offset < loaded_input->n_reads){
                push_count = loaded_input->n_reads - push_offset;
            } 
        }
    }

    *push_counter += push_count;
    push_m->unlock();


    if(push_count == 0){
        tran->n_seqs = 0;
        *actual_push_count = 0;
        FUNC_TIMER_END;
        return;
    }

    // Push inputs to GPU device. ASSUMED MEMCPY DOES NOT FAIL
	resetTransfer(tran);
    tran->n_seqs = push_count;

	copyReads2PinnedMem(loaded_input, tran, push_offset, push_count);
	// at this point, all pointers on tran still point to name, seq, comment, qual addresses on loaded
	// translate reads' addresses to GPU addresses
	convert2DevAddr(tran);
	CUDATransferSeqsIn(tran);

    *actual_push_count = push_count;
    FUNC_TIMER_END;
    return;
}


/**
 * @brief output the previous batch of reads.
 * first transfer device's seqio to host's seq_io.
 * then write from host's seq_io to output
 * 
 * @param first_batch 
 * @param transfer_data 
 */

static void pull(transfer_data_t *tran, double *func_elapsed_ms)
{
    FUNC_TIMER_START;
    if (tran->n_seqs==0){
        FUNC_TIMER_END;
        return;
    }

    // Pull alignment results of the previous batch.
    int total_alns_num;
    hipMemcpy(&total_alns_num, tran->d_total_alns_num, sizeof(int),
            hipMemcpyDeviceToHost);

    int *alnID_offset = (int*)malloc(sizeof(int) * total_alns_num);
    int *alnID_num = (int*)malloc(sizeof(int) * total_alns_num);
    hipMemcpy(alnID_offset, tran->d_alns_offset, sizeof(int) * total_alns_num,
            hipMemcpyDeviceToHost);
    hipMemcpy(alnID_num, tran->d_alns_num, sizeof(int) * total_alns_num,
            hipMemcpyDeviceToHost);


    int *rid = (int*)malloc(sizeof(int) * total_alns_num);
    int64_t *pos = (int64_t*)malloc(sizeof(int64_t) * total_alns_num);

    int total_cigar_len;
    hipMemcpy(&total_cigar_len, tran->d_total_cigar_len, sizeof(int),
            hipMemcpyDeviceToHost);
    int *cigar = (int*)malloc(sizeof(int) * total_cigar_len);
    hipMemcpy(cigar, tran->d_alns_cigar, sizeof(int) * total_cigar_len,
            hipMemcpyDeviceToHost);

    int *cigar_len = (int*)malloc(sizeof(int) * total_alns_num);
    hipMemcpy(cigar_len, tran->d_alns_cigar_len, sizeof(int) * total_alns_num,
            hipMemcpyDeviceToHost);
    uint32_t *cigar_offset = (uint32_t*)malloc(sizeof(uint32_t) * total_alns_num);
    hipMemcpy(cigar_offset, tran->d_alns_cigar_offset, sizeof(uint32_t) * total_alns_num,
            hipMemcpyDeviceToHost);


    // TODO Generate SAM strings.

    FUNC_TIMER_END;
    return;

    // TODO Sequentially write to the output file. -> delegate it to the host.cu level.
}

static void offloader(int gpuid, superbatch_data_t *loadedinput,
        process_data_t *batch_A, transfer_data_t *batch_B,
        int *push_counter, std::mutex *push_m, g3_opt_t *g3_opt)
{
    int this_gpuid;
    if(hipSetDevice(gpuid) != hipSuccess){
       std::cerr << "Offloader for GPU no. " << gpuid 
           << " : hipSetDevice failed" << std::endl;
        return;
    }
    hipGetDevice(&this_gpuid);
    if(this_gpuid != gpuid){
       std::cerr << "Offloader for GPU no. " << gpuid 
           << " : hipSetDevice failed" << std::endl;
        return;
    } 

    double compute_ms, pull_ms, push_ms;
    int push_count, pull_count;
    while(true){
        std::thread t_compute(bwa_align, gpuid, batch_A, g3_opt, &compute_ms);
        pull_count = batch_B->n_seqs;
        pull(batch_B, &pull_ms);
        push(batch_B, loadedinput, push_counter, push_m, g3_opt, 
                &push_count, &push_ms);
        t_compute.join();
        tprof[gpuid][COMPUTE_TOTAL] += (float)compute_ms;
        tprof[gpuid][PULL_TOTAL] += (float)pull_ms;
        tprof[gpuid][PUSH_TOTAL] += (float)push_ms;

        /*
        std::cerr << "* GPU #" << gpuid << " | ";
        std::cerr << std::fixed << std::setprecision(2) << std::setw(8) 
            << compute_ms;
        std::cerr << "ms | GPU-computed " << batch_A->n_seqs << " reads" << std::endl;

        std::cerr << "* GPU #" << gpuid << " | ";
        std::cerr << std::fixed << std::setprecision(2) << std::setw(8) 
            << pull_ms;
        std::cerr << "ms | pulled " << pull_count << " reads (prev. batch)"
            << std::endl;

        std::cerr << "* GPU #" << gpuid << " | ";
        std::cerr << std::fixed << std::setprecision(2) << std::setw(8) 
            << push_ms;
        std::cerr << "ms | pushed " << push_count << " reads (next batch)"
            << std::endl;
            */

        swapData(batch_A, batch_B);
        if(batch_A->n_seqs == 0){
            pull(batch_B, &pull_ms);
            tprof[gpuid][PULL_TOTAL] += (float)pull_ms;

            /*
            std::cerr << "* GPU #" << gpuid << " | ";
            std::cerr << std::fixed << std::setprecision(2) << std::setw(8) 
                << pull_ms;
            std::cerr << "ms | pulled " << pull_count
                << " reads (prev. FINAL batch)" << std::endl;
                */
            break;
        }
    }

    resetTransfer(batch_B);
	return;
}

void aligner(superbatch_data_t *loadedinput, process_data_t *proc[MAX_NUM_GPUS],
        transfer_data_t *tran[MAX_NUM_GPUS], g3_opt_t *g3_opt,
        double *func_elapsed_ms)
{
    FUNC_TIMER_START;
    std::cerr << "* aligning " << loadedinput->n_reads << " reads with " 
        << g3_opt->num_use_gpus << " GPUs" << std::endl;
    if(loadedinput->n_reads == 0){
        FUNC_TIMER_END;
        return;
    }
    std::thread t_offloader[MAX_NUM_GPUS];
    int push_counter = 0;
    std::mutex push_m;


    int gpuid;
    for(gpuid = 0; gpuid < g3_opt->num_use_gpus; gpuid++){
        t_offloader[gpuid] = std::thread(offloader, gpuid, loadedinput,
                proc[gpuid], tran[gpuid], &push_counter, &push_m, g3_opt);
    }
    for(gpuid = 0; gpuid < g3_opt->num_use_gpus; gpuid++){
        t_offloader[gpuid].join();
    }

    FUNC_TIMER_END;
    return;
}
